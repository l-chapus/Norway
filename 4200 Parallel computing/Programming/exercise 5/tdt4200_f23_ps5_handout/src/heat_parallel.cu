#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <unistd.h>
#include <math.h>
#include <sys/time.h>

#include "../inc/argument_utils.h"

// Convert 'struct timeval' into seconds in double prec. floating point
#define WALLTIME(t) ((double)(t).tv_sec + 1e-6 * (double)(t).tv_usec)

typedef int64_t int_t;
typedef double real_t;

int_t
    M,
    N,
    max_iteration,
    snapshot_frequency;

real_t
    *h_temp[2] = { NULL, NULL },
    *h_thermal_diffusivity,
    // TODO 1: Declare device side pointers to store host-side data.
    dt;

#define T(x,y)                      h_temp[0][(y) * (N + 2) + (x)]
#define T_next(x,y)                 h_temp[1][((y) * (N + 2) + (x))]
#define THERMAL_DIFFUSIVITY(x,y)    h_thermal_diffusivity[(y) * (N + 2) + (x)]

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void time_step ( void );
void boundary_condition( void );
void domain_init ( void );
void domain_save ( int_t iteration );
void domain_finalize ( void );


void
swap ( real_t** m1, real_t** m2 )
{
    real_t* tmp;
    tmp = *m1;
    *m1 = *m2;
    *m2 = tmp;
}


int
main ( int argc, char **argv )
{
    OPTIONS *options = parse_args( argc, argv );
    if ( !options )
    {
        fprintf( stderr, "Argument parsing failed\n" );
        exit(1);
    }

    M = options->M;
    N = options->N;
    max_iteration = options->max_iteration;
    snapshot_frequency = options->snapshot_frequency;

    domain_init();

    struct timeval t_start, t_end;
    gettimeofday ( &t_start, NULL );

    for ( int_t iteration = 0; iteration <= max_iteration; iteration++ )
    {
        // TODO 6: Launch the time_step-kernel.

        boundary_condition();

        time_step();

        if ( iteration % snapshot_frequency == 0 )
        {
            printf (
                "Iteration %ld of %ld (%.2lf%% complete)\n",
                iteration,
                max_iteration,
                100.0 * (real_t) iteration / (real_t) max_iteration
            );

            // TODO 8: Copy data from device to host.
            domain_save ( iteration );
        }

        swap( &h_temp[0], &h_temp[1] );
        // TODO 7: Swap device pointers.
    }

    gettimeofday ( &t_end, NULL );
    printf ( "Total elapsed time: %lf seconds\n",
            WALLTIME(t_end) - WALLTIME(t_start)
            );


    domain_finalize();

    exit ( EXIT_SUCCESS );
}


// TODO 4: Make time_step() a CUDA kernel
//         where one thread is responsible for one grid point.
void
time_step ( void )
{
    real_t c, t, b, l, r, K, new_value;

    for ( int_t y = 1; y <= M; y++ )
    {
        for ( int_t x = 1; x <= N; x++ )
        {
            c = T(x, y);

            t = T(x - 1, y);
            b = T(x + 1, y);
            l = T(x, y - 1);
            r = T(x, y + 1);
            K = THERMAL_DIFFUSIVITY(x, y);

            new_value = c + K * dt * ((l - 2 * c + r) + (b - 2 * c + t));

            T_next(x, y) = new_value;
        }
    }
}


// TODO 5: Make boundary_condition() a device function and
//         call it from the time_step-kernel.
//         Chose appropriate threads to set the boundary values.
void
boundary_condition ( void )
{
    for ( int_t x = 1; x <= N; x++ )
    {
        T(x, 0) = T(x, 2);
        T(x, M+1) = T(x, M-1);
    }

    for ( int_t y = 1; y <= M; y++ )
    {
        T(0, y) = T(2, y);
        T(N+1, y) = T(N-1, y);
    }
}


void
domain_init ( void )
{
    h_temp[0] = (real_t*) malloc ( (M+2)*(N+2) * sizeof(real_t) );
    h_temp[1] = (real_t*) malloc ( (M+2)*(N+2) * sizeof(real_t) );
    h_thermal_diffusivity = (real_t*) malloc ( (M+2)*(N+2) * sizeof(real_t) );

    // TODO 2: Allocate device memory.

    dt = 0.1;

    for ( int_t y = 1; y <= M; y++ )
    {
        for ( int_t x = 1; x <= N; x++ )
        {
            real_t temperature = 30 + 30 * sin((x + y) / 20.0);
            real_t diffusivity = 0.05 + (30 + 30 * sin((N - x + y) / 20.0)) / 605.0;

            h_temp[0][ y*(N+2) + x ] = temperature;
            h_temp[1][ y*(N+2) + x ] = temperature;
            h_thermal_diffusivity[ y*(N+2) + x ] = diffusivity;
        }
    }

    // TODO 3: Copy data from host to device.
}


void
domain_save ( int_t iteration )
{
    int_t index = iteration / snapshot_frequency;
    char filename[256];
    memset ( filename, 0, 256*sizeof(char) );
    sprintf ( filename, "data/%.5ld.bin", index );

    FILE *out = fopen ( filename, "wb" );
    if ( ! out ) {
        fprintf(stderr, "Failed to open file: %s\n", filename);
        exit(1);
    }
    for ( int_t iter = 1; iter <= N; iter++)
    {
        fwrite( h_temp[0] + (M+2) * iter + 1, sizeof(real_t), N, out );
    }
    fclose ( out );
}


void
domain_finalize ( void )
{
    free ( h_temp[0] );
    free ( h_temp[1] );
    free ( h_thermal_diffusivity );

    // TODO 9: Free device memory.
}
